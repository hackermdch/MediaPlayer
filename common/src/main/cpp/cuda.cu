#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__ 1
#endif
#include <hip/hip_runtime.h>
#include <>

__device__ __forceinline uint8_t ClampColor(const int v)
{
	if (v <= 0) return 0;
	if (v >= 255) return 255;
	return v;
}

__global__ void Compute(const uint8_t* texY, const uint8_t* texUV, const hipSurfaceObject_t output, const uint32_t width, const uint32_t height, const uint32_t stepY, const uint32_t stepUV)
{
	const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
	const auto idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= width || idy >= height) return;
	const auto indexY = idx + idy * stepY;
	uint32_t indexU, indexV;
	if (idx % 2 == 0)
	{
		indexU = idy / 2 * stepUV + idx;
		indexV = idy / 2 * stepUV + idx + 1;
	}
	else
	{
		indexV = idy / 2 * stepUV + idx;
		indexU = idy / 2 * stepUV + idx - 1;
	}
	const auto y = texY[indexY], u = texUV[indexU], v = texUV[indexV];
	uchar4 color;
	color.x = ClampColor(1.164383 * (y - 16) + 1.596027 * (v - 128));
	color.y = ClampColor(1.164383 * (y - 16) - 0.812968 * (v - 128) - 0.391762 * (u - 128));
	color.z = ClampColor(1.164383 * (y - 16) + 2.017232 * (u - 128));
	color.w = 255;
	surf2Dwrite(color, output, idx * 4, idy);
}

int RunCUDACompute(void* y, void* uv, void* output, void* stream, const uint32_t width, const uint32_t height, const uint32_t stepY, const uint32_t stepUV)
{
	dim3 gs((width + 31) / 32, (height + 31) / 32), bs(32, 32, 1);
	Compute<<<gs, bs, 0, (ihipStream_t*)stream >>>((uint8_t*)y, (uint8_t*)uv, (hipSurfaceObject_t)output, width, height, stepY, stepUV);
	return hipStreamSynchronize((hipStream_t)stream);
}

void* InitCUDA(void* array)
{
	hipResourceDesc resDesc;
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = (hipArray_t)array;
	hipSurfaceObject_t surface;
	hipCreateSurfaceObject(&surface, &resDesc);
	return (void*)surface;
}